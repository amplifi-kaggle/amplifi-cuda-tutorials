#include <stdio.h>
#include <string.h>

#include <sstream>
#include <fstream>
#include <stdlib.h>
#include <iostream>
#include <assert.h>

#include <iostream>
#include <vector>

#include <hipDNN.h>
#include <hipblas.h>

#include <unistd.h>
#include <time.h>
#include <pthread.h>


#ifdef USE_CPP_11
#include <thread>
#endif

#define ASSERT_EQ(A, B) {  \
  if((A)!=(B)) { printf("\n\n[CNMEM FAILED]\n"); this->printCnmemMemoryUsage(); assert(0); }        \
}

#define FatalError(s) {                                                \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;\
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(EXIT_FAILURE);                                                \
}


#define checkCUDNN(status) {                                           \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << status;                           \
      FatalError(_error.str());                                        \
    }                                                                  \
}

#define checkCUBLAS(status) {                                          \
    std::stringstream _error;                                          \
    if (status != HIPBLAS_STATUS_SUCCESS) {                              \
      _error << "CUBLAS failure: " << status;                           \
      FatalError(_error.str());                                        \
    }                                                                  \
}

#define checkCudaErrors(status) {                                      \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      assert(0);                                                        \
      FatalError(_error.str());                                        \
    }                                                                  \
}


inline
hipError_t checkCuda(hipError_t result)
{

  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    //assert(result == hipSuccess);
  }

  return result;
}

#define value_type float
#define DATA_PRECISION  HIPDNN_DATA_FLOAT


/*
typedef enum
{
    HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM         = 0,
    HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM = 1,
    HIPDNN_CONVOLUTION_FWD_ALGO_GEMM                  = 2,
    HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT                = 3,
    HIPDNN_CONVOLUTION_FWD_ALGO_FFT                   = 4,
    HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING            = 5,
    HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD              = 6
} hipdnnConvolutionFwdAlgo_t;
*/
int main(int argc, char **argv)
{
	// first cmdline parameter is used to specify which CNN algorithm to use for derivation
	hipdnnConvolutionFwdAlgo_t fwdAlgo	= (hipdnnConvolutionFwdAlgo_t)atoi(argv[1]);
	
	// create streams
	hipStream_t myStream;
	hipStreamCreate(&myStream);

	// cudnn handle
  hipdnnHandle_t         cudnnHandle;  
  checkCUDNN(hipdnnCreate(&cudnnHandle));

  // link cudnnHandle with this stream
  checkCUDNN(hipdnnSetStream(cudnnHandle, myStream));

	//---------------
	// 1. input fmap
	//---------------
	// NCHW spec for input feature map (fmap) 
	int n_in	= 64;
	int c_in	= 64;
	int h_in	= 224;
	int w_in	= 224;
	// tensor descriptor for input fmap
  hipdnnTensorDescriptor_t       srcTensorDesc;
  checkCUDNN(hipdnnCreateTensorDescriptor(&srcTensorDesc));
  checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n_in, c_in, h_in, w_in));
	// malloc input fmap data array
	void*     srcData;
  hipMalloc(&srcData, n_in*c_in*h_in*w_in*sizeof(value_type));

	//---------------
	// 2-a. filters
	//---------------
	// declare tensor descriptors for CNN layer filters
	int k			= 128;
	int r			= 3;
	int s			= 3;
	// tensor descriptor for filters
  hipdnnFilterDescriptor_t       filterDesc;
  checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
	checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, k, c_in, r, s));
	// malloc filter data array 
  void*     filterData;
  hipMalloc(&filterData, k*c_in*r*s*sizeof(value_type));

	//---------------
	// 2-b. Conv layer spec
	//---------------
	// declare descriptor for the convolution operation to be done within the layer we're testing here
	int pad_h	= 1;
	int pad_w	= 1;
	int stride_h	= 1;
	int stride_w	= 1;
  hipdnnConvolutionDescriptor_t  convDesc;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc, pad_h, pad_w, stride_w, stride_h, 1, 1, HIPDNN_CROSS_CORRELATION));

	//---------------
	// 3. output fmap
	//---------------
	// NCHW spec for output feature map (fmap)
	// find dimension of convolution output
	int n_out, c_out, h_out, w_out;
	checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc, srcTensorDesc, filterDesc, &n_out, &c_out, &h_out, &w_out));
	// declare tensor descriptor for input and output activation maps
  hipdnnTensorDescriptor_t       dstTensorDesc;
	checkCUDNN(hipdnnCreateTensorDescriptor(&dstTensorDesc));
  checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n_out, c_out, h_out, w_out));
	// malloc output fmap data array
	void*			dstData;
  hipMalloc(&dstData, n_out*c_out*h_out*w_out*sizeof(value_type));

	// allocate workspace if required
	std::cout<<"\n-----------------------\n1. fmap and filter size\n-----------------------"<<std::endl;
	std::cout<<" Input  Fmap size: "<<n_in*c_in*h_in*w_in*sizeof(value_type)<<" (bytes)"<<std::endl;
	std::cout<<" Output Fmap size: "<<n_out*c_out*h_out*w_out*sizeof(value_type)<<" (bytes)"<<std::endl;
	std::cout<<" Filter size: "<<k*c_in*r*s*sizeof(value_type)<<" (bytes)\n"<<std::endl;



	// test which cudnn algorithm 
  int requestedAlgoCount = 6; 
  int returnedAlgoCount;

	// profile results
  hipdnnConvolutionFwdAlgoPerf_t*        fwdProfileResults;
  fwdProfileResults = (hipdnnConvolutionFwdAlgoPerf_t*)malloc(sizeof(hipdnnConvolutionFwdAlgoPerf_t)*requestedAlgoCount);
  hipdnnConvolutionFwdAlgoPerf_t *results = fwdProfileResults;

 
	std::cout<<"\n-----------------------\n2. Profile all algorithm\n-----------------------"<<std::endl;
  checkCUDNN(hipdnnFindConvolutionForwardAlgorithm(cudnnHandle, srcTensorDesc, filterDesc, convDesc, dstTensorDesc, requestedAlgoCount, &returnedAlgoCount, results)); 
  for(int algoIndex = 0; algoIndex < returnedAlgoCount; ++algoIndex) {
      printf("^^^^ %s for Algo %d: %f time requiring %llu memory\n", hipdnnGetErrorString(results[algoIndex].status), results[algoIndex].algo, results[algoIndex].time, (unsigned long long)results[algoIndex].memory);
	}
	std::cout<<std::endl;

	std::cout<<"\n-----------------------"<<std::endl;
	printf("Chosen Algorithm: (%d) ",fwdAlgo);
	switch(fwdAlgo) {
		case 0:
			printf("IMPLCIT_GEMM\n");
			break;
		case 1:
			printf("IMPLCIT_PRECOMP_GEMM\n");
			break;
		case 2:
			printf("GEMM\n");
			break;
		case 3:
			printf("DIRECT\n");
			break;
		case 4:
			printf("FFT\n");
			break;
		case 5:
			printf("FFT_TILING\n");
			break;
		case 6:
			printf("WINOGRAD\n");
			break;
		default:
			printf("Invalid algorithm (ERROR)\n");
			assert(0);
	}
	std::cout<<"-----------------------"<<std::endl;

	//---------------
	// 4. Workspace
	//---------------
  // check if workspace is needed for fwdprop
  void* workSpace=NULL;
  size_t sizeInBytes=0;
  // TODO: derive workspace size (search 'hipdnnGetConvolutionForwardWorkspaceSize()')
	hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle, srcTensorDesc, filterDesc, convDesc, dstTensorDesc, fwdAlgo, &sizeInBytes);
	std::cout<<"[Note] WorkingSpace required: "<<sizeInBytes<<" (bytes)"<<std::endl;
	if(sizeInBytes!=0) {
		hipMalloc(&workSpace, sizeInBytes);
	}

	// TIMER
	// Allocate CUDA events that we'll use for timing
	hipEvent_t start, stop;
	// Allocate CUDA events that we'll use for timing
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	// warm-up
  value_type alpha = value_type(1);
  value_type beta  = value_type(0);
	// TODO: forward propagate conv layer (search 'hipdnnConvolutionForward()')
	hipdnnConvolutionForward(cudnnHandle, &alpha, srcTensorDesc, srcData, filterDesc, filterData, convDesc,	fwdAlgo, workSpace, sizeInBytes, &beta, dstTensorDesc, dstData);

  // Record the start event
  checkCudaErrors(hipEventRecord(start, NULL));


	// do forward propagation (conv layer)
  int nIter = 10;
  for (int j = 0; j < nIter; j++)
  {
		// TODO: forward propagate conv layer (search 'hipdnnConvolutionForward()')	
		hipdnnConvolutionForward(
			cudnnHandle, 
			&alpha, 
			srcTensorDesc, 
			srcData, 
			filterDesc, 
			filterData, 
			convDesc,	
			fwdAlgo, 
			workSpace, 
			sizeInBytes, 
			&beta, 
			dstTensorDesc, 
			dstData
		);
	}
  // Record the stop event
  checkCudaErrors(hipEventRecord(stop, NULL));

  // Wait for the stop event to complete
  checkCudaErrors(hipEventSynchronize(stop));

	// free memory for workspace
	if(sizeInBytes!=0) {
		hipFree(workSpace);
	}

  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
  // Compute and print the performance
  float msecPerFwdprop = msecTotal / nIter;
  printf("(CONV layer) Time taken = %.3f (msec)\n",msecPerFwdprop);

	// do forward propagation (activation layer)
	// TODO: create activation descriptor, create and set it up (search 'hipdnnActivationDescriptor_t', hipdnnCreateActivationDescriptor(), and 'hipdnnSetActivationDescriptor()')
	hipdnnActivationDescriptor_t activationDesc;
	hipdnnCreateActivationDescriptor(&activationDesc);
	hipdnnSetActivationDescriptor(activationDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0.0f);

  // Record the start event
  checkCudaErrors(hipEventRecord(start, NULL));
	nIter = 10;
  for (int j = 0; j < nIter; j++)
  {
		// TODO: forward propagate activation layer (search 'hipdnnActivationForward()')
		hipdnnActivationForward(
			cudnnHandle, 
			activationDesc, 
			&alpha, 
			dstTensorDesc, 
			dstData, 
			&beta, 
			dstTensorDesc, 
			dstData
		);
	}
  // Record the stop event
  checkCudaErrors(hipEventRecord(stop, NULL));

  // Wait for the stop event to complete
  checkCudaErrors(hipEventSynchronize(stop));

  msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
  // Compute and print the performance
  msecPerFwdprop = msecTotal / nIter;
  printf("(RELU layer) Time taken = %.3f (msec)\n",msecPerFwdprop);

  return 0;
}
