#include<stdio.h>
#include<hip/hip_runtime.h>

/**
 * CUDA kernel code
 */
__global__
void vectorAdd(float *A,  float *B, float *C, int numElements)
{
	//TODO: derive the indices of A/B/C[] yourself using 'threadIdx.x, blockDim.x, blockIdx.x'
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	//vector addition
	
	//TODO: C = A + B
	if(i < numElements) {
		C[i] = A[i] + B[i];
	}

}

/**
 * Host main routine
 */
int main(void)
{
	hipError_t err = hipSuccess;

	int n = 50000;
	size_t size = n * sizeof(float);
	// alloc host side memory
	//TODO: malloc the host memeory
	float *h_A = (float *)malloc(size);
	float *h_B = (float *)malloc(size);
	float *h_C = (float *)malloc(size);

	//alloc device vetors
	float *d_A = NULL;
	float *d_B = NULL;
	float *d_C = NULL;
	//TODO: use cudaMalloc(void**, int) to allocate device memory
	err = hipMalloc((void **)&d_A, size);
	if(err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));	
	}
	err = hipMalloc((void **)&d_B, size);
	if(err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
	}
	err = hipMalloc((void **)&d_C, size);
	if(err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
	}
	
	//init vector A and vector B
	///TODO: h_A[i] = random number 0 ~ 1 <use rand()>
	for(int i = 0; i < n; i++) {
		h_A[i] = rand() % 2;
		h_B[i] = rand() % 2;
	}

	// copy host data to device
	printf("Copy input vectors to device\n");
	//TODO: use cudaMemcpy(dest, source, size, cudaMemcpyHostToDevice);
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

	//Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads \n", blocksPerGrid, threadsPerBlock);
	//TODO: call vectorAdd function!
	vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);
	err = hipGetLastError();

	if(err != hipSuccess) {
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//Copy device output data to host
	printf("Copy output data to host\n");
	//TODO: cudaMemcpy(dest,source,size,cudaMemcpyDeviceToHost);
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	//Verifiy output
	int pass = 0;
	//TODO: Uncommnet
	
	pass = 1;
	for (int i=0;i<n;i++)
	{
		if(fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
		{
			pass = 0;
			fprintf(stderr, "Result is invalid at element %d!\n",i);
			exit(EXIT_FAILURE);
		}
	}
	
	if (pass)
		printf("Test PASSED\n");
	else
		printf("Test FAILED\n");
	
	//free device memory
	//TODO: use cudaFree();
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	//free host memory
	//TODO: use free();
	free(h_A);
	free(h_B);
	free(h_C);

	printf("Done\n");
	return 0;
}
